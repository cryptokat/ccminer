#include "hip/hip_runtime.h"
static hipStream_t gpustream[MAX_GPUS] = { 0 };
static __constant__ uint64_t c_PaddedMessage80[2]; // padded message (80 bytes + padding)
__constant__ uint64_t precalcvalues[9];
static uint32_t *d_nonce[MAX_GPUS];

#define SWAB32(x)     cuda_swab32(x)

#define R(x, n)       ((x) >> (n))
#define Ch(x, y, z)   ((x & (y ^ z)) ^ z)
#define Maj(x, y, z)  ((x & (y | z)) | (y & z))
#define S0(x)         (ROTR32(x, 2) ^ ROTR32(x, 13) ^ ROTR32(x, 22))
#define S1(x)         (ROTR32(x, 6) ^ ROTR32(x, 11) ^ ROTR32(x, 25))
#define s0(x)         (ROTR32(x, 7) ^ ROTR32(x, 18) ^ R(x, 3))
#define s1(x)         (ROTR32(x, 17) ^ ROTR32(x, 19) ^ R(x, 10))

__constant__ uint32_t sha256_endingTable[] = {
0x80000000, 0x00000000, 0x00000000, 0x00000000, 0x00000000, 0x00000000, 0x00000000, 0x00000000,
0x00000000, 0x00000000, 0x00000000, 0x00000000, 0x00000000, 0x00000000, 0x00000000, 0x00000200,
0x80000000, 0x01400000, 0x00205000, 0x00005088, 0x22000800, 0x22550014, 0x05089742, 0xa0000020,
0x5a880000, 0x005c9400, 0x0016d49d, 0xfa801f00, 0xd33225d0, 0x11675959, 0xf6e6bfda, 0xb30c1549,
0x08b2b050, 0x9d7c4c27, 0x0ce2a393, 0x88e6e1ea, 0xa52b4335, 0x67a16f49, 0xd732016f, 0x4eeb2e91,
0x5dbf55e5, 0x8eee2335, 0xe2bc5ec2, 0xa83f4394, 0x45ad78f7, 0x36f3d0cd, 0xd99c05e8, 0xb0511dc7,
0x69bc7ac4, 0xbd11375b, 0xe3ba71e5, 0x3b209ff2, 0x18feee17, 0xe25ad9e7, 0x13375046, 0x0515089d,
0x4f0d0f04, 0x2627484e, 0x310128d2, 0xc668b434, 0x420841cc, 0x62d311b8, 0xe59ba771, 0x85a7a484
};

__constant__ uint32_t sha256_constantTable[64] = {
0x428a2f98, 0x71374491, 0xb5c0fbcf, 0xe9b5dba5, 0x3956c25b, 0x59f111f1, 0x923f82a4, 0xab1c5ed5,
0xd807aa98, 0x12835b01, 0x243185be, 0x550c7dc3, 0x72be5d74, 0x80deb1fe, 0x9bdc06a7, 0xc19bf174,
0xe49b69c1, 0xefbe4786, 0x0fc19dc6, 0x240ca1cc, 0x2de92c6f, 0x4a7484aa, 0x5cb0a9dc, 0x76f988da,
0x983e5152, 0xa831c66d, 0xb00327c8, 0xbf597fc7, 0xc6e00bf3, 0xd5a79147, 0x06ca6351, 0x14292967,
0x27b70a85, 0x2e1b2138, 0x4d2c6dfc, 0x53380d13, 0x650a7354, 0x766a0abb, 0x81c2c92e, 0x92722c85,
0xa2bfe8a1, 0xa81a664b, 0xc24b8b70, 0xc76c51a3, 0xd192e819, 0xd6990624, 0xf40e3585, 0x106aa070,
0x19a4c116, 0x1e376c08, 0x2748774c, 0x34b0bcb5, 0x391c0cb3, 0x4ed8aa4a, 0x5b9cca4f, 0x682e6ff3,
0x748f82ee, 0x78a5636f, 0x84c87814, 0x8cc70208, 0x90befffa, 0xa4506ceb, 0xbef9a3f7, 0xc67178f2
};


#define TFBIG_KINIT(k0, k1, k2, k3, k4, k5, k6, k7, k8, t0, t1, t2) { \
		k8 = k0 ^ k1 ^ k2 ^ k3 ^ k4 ^ k5 ^ k6 ^ k7 ^ make_uint2(0xA9FC1A22UL, 0x1BD11BDA); \
		t2 = t0 ^ t1; \
	}

#define TFBIG_MIX8(w0, w1, w2, w3, w4, w5, w6, w7, rc0, rc1, rc2, rc3) { \
		TFBIG_MIX(w0, w1, rc0); \
		TFBIG_MIX(w2, w3, rc1); \
		TFBIG_MIX(w4, w5, rc2); \
		TFBIG_MIX(w6, w7, rc3); \
	}

#define TFBIG_4e(s)  { \
		TFBIG_ADDKEY(p[0], p[1], p[2], p[3], p[4], p[5], p[6], p[7], h, t, s); \
		TFBIG_MIX8(p[0], p[1], p[2], p[3], p[4], p[5], p[6], p[7], 46, 36, 19, 37); \
		TFBIG_MIX8(p[2], p[1], p[4], p[7], p[6], p[5], p[0], p[3], 33, 27, 14, 42); \
		TFBIG_MIX8(p[4], p[1], p[6], p[3], p[0], p[5], p[2], p[7], 17, 49, 36, 39); \
		TFBIG_MIX8(p[6], p[1], p[0], p[7], p[2], p[5], p[4], p[3], 44,  9, 54, 56); \
	}

#define TFBIG_4o(s)  { \
		TFBIG_ADDKEY(p[0], p[1], p[2], p[3], p[4], p[5], p[6], p[7], h, t, s); \
		TFBIG_MIX8(p[0], p[1], p[2], p[3], p[4], p[5], p[6], p[7], 39, 30, 34, 24); \
		TFBIG_MIX8(p[2], p[1], p[4], p[7], p[6], p[5], p[0], p[3], 13, 50, 10, 17); \
		TFBIG_MIX8(p[4], p[1], p[6], p[3], p[0], p[5], p[2], p[7], 25, 29, 39, 43); \
		TFBIG_MIX8(p[6], p[1], p[0], p[7], p[2], p[5], p[4], p[3],  8, 35, 56, 22); \
	}

#define TFBIG_MIX_PRE(x0, x1, rc) { \
		x0 = x0 + x1; \
		x1 = ROTL64(x1, rc) ^ x0; \
				}

#define TFBIG_MIX8_UI2(w0, w1, w2, w3, w4, w5, w6, w7, rc0, rc1, rc2, rc3) { \
		TFBIG_MIX_UI2(w0, w1, rc0); \
		TFBIG_MIX_UI2(w2, w3, rc1); \
		TFBIG_MIX_UI2(w4, w5, rc2); \
		TFBIG_MIX_UI2(w6, w7, rc3); \
		}

#define TFBIG_MIX8_PRE(w0, w1, w2, w3, w4, w5, w6, w7, rc0, rc1, rc2, rc3) { \
		TFBIG_MIX_PRE(w0, w1, rc0); \
		TFBIG_MIX_PRE(w2, w3, rc1); \
		TFBIG_MIX_PRE(w4, w5, rc2); \
		TFBIG_MIX_PRE(w6, w7, rc3); \
				}

#define TFBIG_4e_UI2(s)  { \
		TFBIG_ADDKEY_UI2(p[0], p[1], p[2], p[3], p[4], p[5], p[6], p[7], h, t, s); \
		TFBIG_MIX8_UI2(p[0], p[1], p[2], p[3], p[4], p[5], p[6], p[7], 46, 36, 19, 37); \
		TFBIG_MIX8_UI2(p[2], p[1], p[4], p[7], p[6], p[5], p[0], p[3], 33, 27, 14, 42); \
		TFBIG_MIX8_UI2(p[4], p[1], p[6], p[3], p[0], p[5], p[2], p[7], 17, 49, 36, 39); \
		TFBIG_MIX8_UI2(p[6], p[1], p[0], p[7], p[2], p[5], p[4], p[3], 44,  9, 54, 56); \
		}

#define TFBIG_4e_PRE(s)  { \
		TFBIG_ADDKEY_PRE(p[0], p[1], p[2], p[3], p[4], p[5], p[6], p[7], h, t, s); \
		TFBIG_MIX8_PRE(p[0], p[1], p[2], p[3], p[4], p[5], p[6], p[7], 46, 36, 19, 37); \
		TFBIG_MIX8_PRE(p[2], p[1], p[4], p[7], p[6], p[5], p[0], p[3], 33, 27, 14, 42); \
		TFBIG_MIX8_PRE(p[4], p[1], p[6], p[3], p[0], p[5], p[2], p[7], 17, 49, 36, 39); \
		TFBIG_MIX8_PRE(p[6], p[1], p[0], p[7], p[2], p[5], p[4], p[3], 44,  9, 54, 56); \
				}

#define TFBIG_4o_UI2(s)  { \
		TFBIG_ADDKEY_UI2(p[0], p[1], p[2], p[3], p[4], p[5], p[6], p[7], h, t, s); \
		TFBIG_MIX8_UI2(p[0], p[1], p[2], p[3], p[4], p[5], p[6], p[7], 39, 30, 34, 24); \
		TFBIG_MIX8_UI2(p[2], p[1], p[4], p[7], p[6], p[5], p[0], p[3], 13, 50, 10, 17); \
		TFBIG_MIX8_UI2(p[4], p[1], p[6], p[3], p[0], p[5], p[2], p[7], 25, 29, 39, 43); \
		TFBIG_MIX8_UI2(p[6], p[1], p[0], p[7], p[2], p[5], p[4], p[3],  8, 35, 56, 22); \
		}

#define TFBIG_4o_PRE(s)  { \
		TFBIG_ADDKEY_PRE(p[0], p[1], p[2], p[3], p[4], p[5], p[6], p[7], h, t, s); \
		TFBIG_MIX8_PRE(p[0], p[1], p[2], p[3], p[4], p[5], p[6], p[7], 39, 30, 34, 24); \
		TFBIG_MIX8_PRE(p[2], p[1], p[4], p[7], p[6], p[5], p[0], p[3], 13, 50, 10, 17); \
		TFBIG_MIX8_PRE(p[4], p[1], p[6], p[3], p[0], p[5], p[2], p[7], 25, 29, 39, 43); \
		TFBIG_MIX8_PRE(p[6], p[1], p[0], p[7], p[2], p[5], p[4], p[3],  8, 35, 56, 22); \
		}


static void precalc(int thr_id, uint64_t *PaddedMessage)
{
    uint64_t h0, h1, h2, h3, h4, h5, h6, h7, h8;
    uint64_t t0, t1, t2;

    h0 = 0x4903ADFF749C51CEull;
    h1 = 0x0D95DE399746DF03ull;
    h2 = 0x8FD1934127C79BCEull;
    h3 = 0x9A255629FF352CB1ull;
    h4 = 0x5DB62599DF6CA7B0ull;
    h5 = 0xEABE394CA9D5C3F4ull;
    h6 = 0x991112C71A75B523ull;
    h7 = 0xAE18A40B660FCC33ull;
    h8 = h0 ^ h1 ^ h2 ^ h3 ^ h4 ^ h5 ^ h6 ^ h7 ^ SPH_C64(0x1BD11BDAA9FC1A22);

    t0 = 64; // ptr
    t1 = 0x7000000000000000ull;
    t2 = 0x7000000000000040ull;

    uint64_t p[8];
    for (int i = 0; i<8; i++)
        p[i] = PaddedMessage[i];

    TFBIG_4e_PRE(0);
    TFBIG_4o_PRE(1);
    TFBIG_4e_PRE(2);
    TFBIG_4o_PRE(3);
    TFBIG_4e_PRE(4);
    TFBIG_4o_PRE(5);
    TFBIG_4e_PRE(6);
    TFBIG_4o_PRE(7);
    TFBIG_4e_PRE(8);
    TFBIG_4o_PRE(9);
    TFBIG_4e_PRE(10);
    TFBIG_4o_PRE(11);
    TFBIG_4e_PRE(12);
    TFBIG_4o_PRE(13);
    TFBIG_4e_PRE(14);
    TFBIG_4o_PRE(15);
    TFBIG_4e_PRE(16);
    TFBIG_4o_PRE(17);
    TFBIG_ADDKEY_PRE(p[0], p[1], p[2], p[3], p[4], p[5], p[6], p[7], h, t, 18);

    uint64_t buffer[9];

    buffer[0] = PaddedMessage[0] ^ p[0];
    buffer[1] = PaddedMessage[1] ^ p[1];
    buffer[2] = PaddedMessage[2] ^ p[2];
    buffer[3] = PaddedMessage[3] ^ p[3];
    buffer[4] = PaddedMessage[4] ^ p[4];
    buffer[5] = PaddedMessage[5] ^ p[5];
    buffer[6] = PaddedMessage[6] ^ p[6];
    buffer[7] = PaddedMessage[7] ^ p[7];
    buffer[8] = t2;
    CUDA_SAFE_CALL(hipMemcpyToSymbolAsync(HIP_SYMBOL(precalcvalues), buffer, sizeof(buffer), 0, hipMemcpyHostToDevice, gpustream[thr_id]));
}